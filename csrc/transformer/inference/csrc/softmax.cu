#include "hip/hip_runtime.h"
/*
Copyright 2022 The Microsoft DeepSpeed Team
*/

#include <limits>
#include "inference_cuda_layers.h"

#include <hip/hip_runtime_api.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>

#define ATTN_THREADS 1024
#define MAX_REG_SIZE 8

#define Attn_Threads_111 128
#define Reduce_Threads 32
#define attn_warps 4
#define MAX_ATTN_REG 4  // MAX Head Size 256

#define minus_infinity -10000.0

void CheckCudaErrorAux(const char* file, unsigned line)
{
    hipError_t err = hipGetLastError();
    if (err == hipSuccess) return;
    std::cerr << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line
              << std::endl;
    throw std::runtime_error("CUDA ERROR!!!\n");
}

#define CUDA_CHECK_ERROR() CheckCudaErrorAux(__FILE__, __LINE__)

namespace cg = cooperative_groups;

__global__ void attn_softmax_v2(__half* vals,
                                __half* mask,
                                __half* alibi,
                                float layer_scale,
                                bool triangular,
                                bool recompute,
                                bool local_attention,
                                int window_size,
                                int total_count,
                                int heads,
                                int sequence_length,
                                int num_seq,
                                int head_offset,
                                int mask_stride,
                                int mp_size,
                                int iterations,
                                int reduceWidth)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    float2 low_data[MAX_REG_SIZE];
    float2 high_data[MAX_REG_SIZE];
    const __half zero_h = __float2half(0.f);

    int wid = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;
    int warp_num = blockDim.x >> 5;

    int reduce_blocks = reduceWidth >> 5;
    int seq_lane = threadIdx.x % reduceWidth;

    __shared__ float partialSum[MAX_WARP_NUM];

    int iter_offset = blockIdx.x * (warp_num / reduce_blocks) + (wid / reduce_blocks);
    int batch_idx = iter_offset / (num_seq * heads);
    int alibi_offset = batch_idx * heads * mp_size + head_offset;
    int mask_offset = batch_idx * mask_stride + (iter_offset % mask_stride);

    if (iter_offset < total_count) {
        vals += (iter_offset * sequence_length);

        alibi_offset = (alibi_offset + ((iter_offset / num_seq) % heads)) * sequence_length;
        mask_offset = mask_offset * sequence_length;
        int seq_id = iter_offset % num_seq;
        int seq_id4 = seq_id >> 2;

        int real_seq_id = seq_id + (num_seq == sequence_length ? 0 : sequence_length);
        int window_stride4 = (local_attention && (real_seq_id >> 2) > (window_size >> 2))
                                 ? (real_seq_id >> 2) - (window_size >> 2)
                                 : 0;
        int window_stride =
            (local_attention && real_seq_id >= window_size) ? real_seq_id - window_size : -1;

        float max_val = minus_infinity;
        // if (lane == 0) printf("%d, %d: %d \n", wid, blockIdx.x, mask_offset);
        for (int i = 0; i < iterations; i++) {
            int data_id = i * (reduceWidth << 2) + (seq_lane << 2);
            if ((!triangular || ((data_id >> 2) <= seq_id4)) && (data_id >> 2) >= window_stride4 &&
                data_id < sequence_length) {
                if ((sequence_length - data_id) >= 4) {
                    low_data[i].x = data_id > window_stride
                                        ? __half2float(vals[data_id]) * layer_scale
                                        : minus_infinity;
                    low_data[i].y = ((!triangular || ((data_id + 1) <= seq_id)) &&
                                     (data_id + 1) > window_stride)
                                        ? __half2float(vals[data_id + 1]) * layer_scale
                                        : minus_infinity;
                    high_data[i].x = ((!triangular || ((data_id + 2) <= seq_id)) &&
                                      (data_id + 2) > window_stride)
                                         ? __half2float(vals[data_id + 2]) * layer_scale
                                         : minus_infinity;
                    high_data[i].y = ((!triangular || ((data_id + 3) <= seq_id)) &&
                                      (data_id + 3) > window_stride)
                                         ? __half2float(vals[data_id + 3]) * layer_scale
                                         : minus_infinity;
                    if (alibi) {
                        low_data[i].x = low_data[i].x + __half2float(alibi[data_id + alibi_offset]);
                        low_data[i].y =
                            low_data[i].y + __half2float(alibi[data_id + alibi_offset + 1]);
                        high_data[i].x =
                            high_data[i].x + __half2float(alibi[data_id + alibi_offset + 2]);
                        high_data[i].y =
                            high_data[i].y + __half2float(alibi[data_id + alibi_offset + 3]);
                    }
                    if (mask) {
                        low_data[i].x += __half2float(mask[data_id + mask_offset]);
                        low_data[i].y += __half2float(mask[data_id + mask_offset + 1]);
                        high_data[i].x += __half2float(mask[data_id + mask_offset + 2]);
                        high_data[i].y += __half2float(mask[data_id + mask_offset + 3]);
                    }
                } else {
                    low_data[i].x = data_id > window_stride
                                        ? __half2float(vals[data_id]) * layer_scale
                                        : minus_infinity;
                    low_data[i].y = (((!triangular || (data_id + 1) <= seq_id) &&
                                      (data_id + 1) > window_stride) &&
                                     (data_id + 1) < sequence_length)
                                        ? __half2float(vals[data_id + 1]) * layer_scale
                                        : minus_infinity;
                    high_data[i].x = (((!triangular || (data_id + 2) <= seq_id) &&
                                       (data_id + 2) > window_stride) &&
                                      (data_id + 2) < sequence_length)
                                         ? __half2float(vals[data_id + 2]) * layer_scale
                                         : minus_infinity;
                    if (alibi) {
                        low_data[i].x = low_data[i].x + __half2float(alibi[data_id + alibi_offset]);
                        if ((data_id + 1) < sequence_length)
                            low_data[i].y =
                                low_data[i].y + __half2float(alibi[data_id + alibi_offset + 1]);
                        if ((data_id + 2) < sequence_length)
                            high_data[i].x =
                                high_data[i].x + __half2float(alibi[data_id + alibi_offset + 2]);
                    }
                    high_data[i].y = minus_infinity;
                    if (mask) {
                        low_data[i].x += __half2float(mask[data_id + mask_offset]);
                        if ((data_id + 1) < sequence_length)
                            low_data[i].y += __half2float(mask[data_id + mask_offset + 1]);
                        if ((data_id + 2) < sequence_length)
                            high_data[i].x += __half2float(mask[data_id + mask_offset + 2]);
                    }
                }
                max_val = (low_data[i].x > max_val ? low_data[i].x : max_val);
                max_val = (low_data[i].y > max_val ? low_data[i].y : max_val);
                max_val = (high_data[i].x > max_val ? high_data[i].x : max_val);
                max_val = (high_data[i].y > max_val ? high_data[i].y : max_val);
            } else {
                low_data[i].x = minus_infinity;
                low_data[i].y = minus_infinity;
                high_data[i].x = minus_infinity;
                high_data[i].y = minus_infinity;
            }
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) {
            auto temp = g.shfl_xor(max_val, i);
            max_val = (temp > max_val ? temp : max_val);
        }

        if (reduceWidth > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = max_val;
            b.sync();

            if (lane < warp_num) max_val = partialSum[lane];

            b.sync();

            for (int i = 1; i < reduce_blocks; i *= 2) {
                auto temp = g.shfl_xor(max_val, i);
                max_val = (temp > max_val ? temp : max_val);
            }

            max_val = g.shfl(max_val, threadIdx.x / WARP_SIZE);
        }
        float sum = 0;
        for (int i = 0; i < iterations; i++) {
            low_data[i].x = __expf(low_data[i].x - max_val);
            low_data[i].y = __expf(low_data[i].y - max_val);
            high_data[i].x = __expf(high_data[i].x - max_val);
            high_data[i].y = __expf(high_data[i].y - max_val);

            sum += (low_data[i].x + low_data[i].y + high_data[i].x + high_data[i].y);
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) sum += g.shfl_xor(sum, i);

        if (reduceWidth > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = sum;
            b.sync();

            if (lane < warp_num) sum = partialSum[lane];

            b.sync();

            for (int i = 1; i < reduce_blocks; i *= 2) { sum += g.shfl_xor(sum, i); }

            sum = g.shfl(sum, threadIdx.x / WARP_SIZE);
        }
        sum += 1e-6;
        for (int i = 0; i < iterations; i++) {
            int data_id = i * (reduceWidth << 2) + (seq_lane << 2);

            if (data_id < sequence_length) {
                if ((sequence_length - data_id) >= 4) {
                    vals[data_id] = __float2half(low_data[i].x / sum);
                    vals[data_id + 1] = __float2half(low_data[i].y / sum);
                    vals[data_id + 2] = __float2half(high_data[i].x / sum);
                    vals[data_id + 3] = __float2half(high_data[i].y / sum);
                } else {
                    vals[data_id] = __float2half(low_data[i].x / sum);
                    if ((data_id + 1) < sequence_length)
                        vals[data_id + 1] = __float2half(low_data[i].y / sum);
                    if ((data_id + 2) < sequence_length)
                        vals[data_id + 2] = __float2half(high_data[i].x / sum);
                }
            }
        }
    }
}

__global__ void attn_softmax_v2(float* vals,
                                float* attn_mask,
                                float* alibi,
                                float layer_scale,
                                bool triangular,
                                bool recompute,
                                bool local_attention,
                                int window_size,
                                int total_count,
                                int heads,
                                int sequence_length,
                                int num_seq,
                                int head_offset,
                                int mask_stride,
                                int mp_size,
                                int iterations,
                                int reduceWidth)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    float4 data[MAX_REG_SIZE];

    int wid = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;
    int warp_num = blockDim.x >> 5;

    int reduce_blocks = reduceWidth >> 5;
    int seq_lane = threadIdx.x % reduceWidth;

    __shared__ float partialSum[MAX_WARP_NUM];

    int iter_offset = blockIdx.x * (warp_num / reduce_blocks) + (wid / reduce_blocks);
    if (iter_offset < total_count) {
        vals += (iter_offset * sequence_length);

        int batch_idx = iter_offset / (num_seq * heads);
        int alibi_offset = batch_idx * heads * mp_size + head_offset;
        int mask_offset = batch_idx * mask_stride + (iter_offset % mask_stride);
        mask_offset = mask_offset * sequence_length;
        int seq_id = iter_offset % num_seq;
        int seq_id4 = seq_id >> 2;

        int real_seq_id = seq_id + (num_seq == sequence_length ? 0 : sequence_length);
        int window_stride4 = (local_attention && (real_seq_id >> 2) > (window_size >> 2))
                                 ? (real_seq_id >> 2) - (window_size >> 2)
                                 : 0;
        int window_stride =
            (local_attention && real_seq_id >= window_size) ? real_seq_id - window_size : -1;

        float max_val = minus_infinity;

        for (int i = 0; i < iterations; i++) {
            int data_id = i * (reduceWidth << 2) + (seq_lane << 2);
            if ((!triangular || ((data_id >> 2) <= seq_id4)) && (data_id >> 2) >= window_stride4 &&
                data_id < sequence_length) {
                if ((sequence_length - data_id) >= 4) {
                    data[i].x = (data_id > window_stride ? vals[data_id] : minus_infinity);
                    data[i].y = ((!triangular || ((data_id + 1) <= seq_id)) &&
                                 (data_id + 1) > window_stride)
                                    ? vals[data_id + 1]
                                    : minus_infinity;
                    data[i].z = ((!triangular || ((data_id + 2) <= seq_id)) &&
                                 (data_id + 2) > window_stride)
                                    ? vals[data_id + 2]
                                    : minus_infinity;
                    data[i].w = ((!triangular || ((data_id + 3) <= seq_id)) &&
                                 (data_id + 3) > window_stride)
                                    ? vals[data_id + 3]
                                    : minus_infinity;
                    if (attn_mask) {
                        data[i].x += attn_mask[data_id + mask_offset];
                        data[i].y += attn_mask[data_id + mask_offset + 1];
                        data[i].z += attn_mask[data_id + mask_offset + 2];
                        data[i].w += attn_mask[data_id + mask_offset + 3];
                    }
                } else {
                    data[i].x = data_id > window_stride ? vals[data_id] : minus_infinity;
                    data[i].y = (((!triangular || (data_id + 1) <= seq_id)) &&
                                 (data_id + 1) > window_stride && (data_id + 1) < sequence_length)
                                    ? (vals[data_id + 1])
                                    : minus_infinity;
                    data[i].z = (((!triangular || (data_id + 2) <= seq_id)) &&
                                 (data_id + 2) > window_stride && (data_id + 2) < sequence_length)
                                    ? (vals[data_id + 2])
                                    : minus_infinity;
                    data[i].w = minus_infinity;
                    if (attn_mask) {
                        data[i].x += attn_mask[data_id + mask_offset];
                        if ((data_id + 1) < sequence_length)
                            data[i].y += attn_mask[data_id + mask_offset + 1];
                        if ((data_id + 2) < sequence_length)
                            data[i].z += attn_mask[data_id + mask_offset + 2];
                    }
                }
                max_val = (data[i].x > max_val ? data[i].x : max_val);
                max_val = (data[i].y > max_val ? data[i].y : max_val);
                max_val = (data[i].z > max_val ? data[i].z : max_val);
                max_val = (data[i].w > max_val ? data[i].w : max_val);
            } else {
                data[i].x = minus_infinity;
                data[i].y = minus_infinity;
                data[i].z = minus_infinity;
                data[i].w = minus_infinity;
            }
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) {
            auto temp = g.shfl_xor(max_val, i);
            max_val = (temp > max_val ? temp : max_val);
        }

        if (reduceWidth > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = max_val;
            b.sync();
            if (lane < warp_num) max_val = partialSum[lane];
            b.sync();
            for (int i = 1; i < reduce_blocks; i *= 2) {
                auto temp = g.shfl_xor(max_val, i);
                max_val = (temp > max_val ? temp : max_val);
            }
            max_val = g.shfl(max_val, threadIdx.x / WARP_SIZE);
        }

        float sum = 0;
        for (int i = 0; i < iterations; i++) {
            data[i].x = __expf(data[i].x - max_val);
            data[i].y = __expf(data[i].y - max_val);
            data[i].z = __expf(data[i].z - max_val);
            data[i].w = __expf(data[i].w - max_val);

            sum += (data[i].x + data[i].y + data[i].z + data[i].w);
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) sum += g.shfl_xor(sum, i);

        if (reduceWidth > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = sum;
            b.sync();
            if (lane < warp_num) sum = partialSum[lane];
            b.sync();
            for (int i = 1; i < reduce_blocks; i *= 2) { sum += g.shfl_xor(sum, i); }
            sum = g.shfl(sum, threadIdx.x / WARP_SIZE);
        }
        sum += 1e-6;

        for (int i = 0; i < iterations; i++) {
            int data_id = i * (reduceWidth << 2) + (seq_lane << 2);

            if (data_id < sequence_length) {
                if ((sequence_length - data_id) >= 4) {
                    vals[data_id] = data[i].x / sum;
                    vals[data_id + 1] = data[i].y / sum;
                    vals[data_id + 2] = data[i].z / sum;
                    vals[data_id + 3] = data[i].w / sum;
                } else {
                    vals[data_id] = data[i].x / sum;
                    if ((data_id + 1) < sequence_length) vals[data_id + 1] = data[i].y / sum;
                    if ((data_id + 2) < sequence_length) vals[data_id + 2] = data[i].z / sum;
                }
            }
        }
    }
}

template <typename T>
void launch_attn_softmax_v2(T* vals,
                            T* mask,
                            T* alibi,
                            float layer_scale,
                            bool triangular,
                            bool recompute,
                            bool local_attention,
                            int window_size,
                            int batch_size,
                            int heads,
                            int num_seq,
                            int sequence_length,
                            int head_offset,
                            int mask_stride,
                            int mp_size,
                            hipStream_t stream)
{
    int total_count = batch_size * heads * num_seq;
    dim3 grid_dim((total_count - 1) / (WARP_SIZE / ((sequence_length - 1) / ATTN_THREADS + 1)) + 1);
    dim3 block_dim(ATTN_THREADS);

    const int reduce_width = ((sequence_length - 1) / ATTN_THREADS + 1) * WARP_SIZE;
    const int iterations = (sequence_length - 1) / (reduce_width << 2) + 1;

    if (sequence_length <= 32768)
        attn_softmax_v2<<<grid_dim, block_dim, 0, stream>>>(vals,
                                                            mask,
                                                            alibi,
                                                            layer_scale,
                                                            triangular,
                                                            recompute,
                                                            local_attention,
                                                            window_size,
                                                            total_count,
                                                            heads,
                                                            sequence_length,
                                                            num_seq,
                                                            head_offset,
                                                            mask_stride,
                                                            mp_size,
                                                            iterations,
                                                            reduce_width);
    else
        throw std::runtime_error("Unsupport Seq_Length!");
}

template void launch_attn_softmax_v2(float* vals,
                                     float* mask,
                                     float* alibi,
                                     float layer_scale,
                                     bool triangular,
                                     bool recompute,
                                     bool local_attention,
                                     int window_size,
                                     int batch_size,
                                     int heads,
                                     int num_seq,
                                     int sequence_length,
                                     int head_offset,
                                     int mask_stride,
                                     int mp_size,
                                     hipStream_t stream);
template void launch_attn_softmax_v2(__half* vals,
                                     __half* mask,
                                     __half* alibi,
                                     float layer_scale,
                                     bool triangular,
                                     bool recompute,
                                     bool local_attention,
                                     int window_size,
                                     int batch_size,
                                     int heads,
                                     int num_seq,
                                     int sequence_length,
                                     int head_offset,
                                     int mask_stride,
                                     int mp_size,
                                     hipStream_t stream);

__device__ void attn_score(__half* shared_soft,
                           __half* query,
                           __half* key_merged,
                           __half* attn_bias,
                           bool merging,
                           float norm_factor,
                           int inp_size,
                           int total_count,
                           int num_seq,
                           int hidden,
                           int value_length)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int wid = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;
    int warp_num = blockDim.x >> 5;
    bool is_prompt = (value_length == num_seq);
    __half2 queries_low[MAX_ATTN_REG];
    __half2 queries_high[MAX_ATTN_REG];

    float2* query_cast = reinterpret_cast<float2*>(query);
    float2* key_cast = reinterpret_cast<float2*>(is_prompt ? query + (hidden << 1) : key_merged);
    float2* bias_cast = reinterpret_cast<float2*>(attn_bias);

    float2* key_merged_cast = reinterpret_cast<float2*>(key_merged);
    float2* new_key_cast = reinterpret_cast<float2*>(query + (hidden << 1));

    __half2 norm_factor_h = __float2half2_rn(norm_factor);

    int input_offset = (blockIdx.x * warp_num + wid);
    int hidden31 = is_prompt ? (hidden >> 1) * 3 : (hidden >> 1);
    if (input_offset < total_count) {
        query_cast +=
            (input_offset % num_seq) * (hidden >> 1) * 3 + (input_offset / num_seq) * inp_size;
        bias_cast += (input_offset / num_seq) * inp_size;
        int row = lane;
        int p = 0;

        while (row < inp_size) {
            float2 querie = query_cast[row];

            __half2* query_value = reinterpret_cast<__half2*>(&querie);

            if (attn_bias) {
                float2 bias_reg = bias_cast[row];
                __half2* bias_value = reinterpret_cast<__half2*>(&bias_reg);
                queries_low[p] = query_value[0] + bias_value[0];
                queries_high[p] = query_value[1] + bias_value[1];
            } else {
                queries_low[p] = query_value[0] * norm_factor_h;
                queries_high[p] = query_value[1] * norm_factor_h;
            }

            p++;
            row += WARP_SIZE;
        }

        int seq_key = input_offset / num_seq;

        key_cast += (seq_key * inp_size);
        if (key_merged != nullptr) key_merged_cast += (seq_key * inp_size);

        bias_cast += (hidden >> 1);
        int key_size = total_count / num_seq;
        int score_index = 0;

        if (seq_key < key_size) {
            {
                float scores[WARP_SIZE];
                int warp_iter;
                for (int i = 0; i < value_length; i += WARP_SIZE) {
                    warp_iter = (value_length - i) > WARP_SIZE ? WARP_SIZE : (value_length - i);
#pragma unroll
                    for (int p = 0; p < warp_iter; p++) { scores[p] = 0; }

                    for (int k = 0; k < warp_iter; k++) {
                        row = lane;
                        int p = 0;
                        while (row < inp_size) {
                            float2 key_value_reg = key_cast[row];
                            if (is_prompt && (key_merged != nullptr) &&
                                (input_offset % num_seq) == 0)
                                key_merged_cast[row] = key_value_reg;
                            __half2* key_value = reinterpret_cast<__half2*>(&key_value_reg);

                            if (attn_bias) {
                                float2 bias_reg = bias_cast[row];
                                __half2* bias_value = reinterpret_cast<__half2*>(&bias_reg);
                                key_value[0] += bias_value[0];
                                key_value[1] += bias_value[1];
                            }
                            key_value[0] *= norm_factor_h;
                            key_value[1] *= norm_factor_h;

                            float2 mul[2];
                            mul[0] = __half22float2(queries_low[p] * key_value[0]);
                            mul[1] = __half22float2(queries_high[p] * key_value[1]);
                            scores[k] = (mul[0].x + mul[0].y) + (mul[1].x + mul[1].y);
                            row += WARP_SIZE;
                            p++;
                        }
                        key_cast += hidden31;  //(hidden >> 1);
                        if (is_prompt && (key_merged != nullptr) && (input_offset % num_seq) == 0)
                            key_merged_cast += (hidden >> 1);
#pragma unroll
                        for (int w = 1; w < WARP_SIZE; w *= 2)
                            scores[k] += g.shfl_xor(scores[k], w);
                    }
                    if (lane < (warp_iter >> 1)) {
                        shared_soft[wid * 1024 + ((lane + score_index) << 1)] =
                            __float2half(scores[(lane << 1)]);
                        shared_soft[wid * 1024 + ((lane + score_index) << 1) + 1] =
                            __float2half(scores[(lane << 1) + 1]);
                    }
                    score_index += (warp_iter >> 1);
                }
                if (warp_iter % 2 == 1) {
                    if (lane == (warp_iter >> 1)) {
                        shared_soft[wid * 1024 + (score_index << 1)] =
                            __float2half(scores[(lane << 1)]);
                    }
                }
            }
            if (!is_prompt && key_merged != nullptr) {
                new_key_cast += ((input_offset / num_seq) * inp_size);

                key_merged_cast = reinterpret_cast<float2*>(key_merged);
                key_merged_cast +=
                    (input_offset / num_seq) * inp_size + ((hidden >> 1) * value_length);

                row = lane;
                int p = 0;
                float score = 0;
                while (row < inp_size) {
                    float2 new_key_data = new_key_cast[row];
                    if ((input_offset % num_seq) == 0) key_merged_cast[row] = new_key_data;
                    __half2* key_value = reinterpret_cast<__half2*>(&new_key_data);

                    key_value[0] *= norm_factor_h;
                    key_value[1] *= norm_factor_h;

                    float2 mul[2];
                    mul[0] = __half22float2(queries_low[p] * key_value[0]);
                    mul[1] = __half22float2(queries_high[p] * key_value[1]);
                    score += (mul[0].x + mul[0].y) + (mul[1].x + mul[1].y);
                    row += WARP_SIZE;
                    p++;
                }
#pragma unroll
                for (int w = 1; w < WARP_SIZE; w *= 2) score += g.shfl_down(score, w);

                if (lane == 0) {
                    if ((value_length + 1) % 2 == 0) {
                        shared_soft[wid * 1024 + (score_index << 1) + 1] = __float2half(score);
                    } else {
                        shared_soft[wid * 1024 + (score_index << 1)] = __float2half(score);
                    }
                }
            }
        }
    }
}

__device__ void attn_score(float* shared_soft,
                           float* query,
                           float* key_merged,
                           float* attn_bias,
                           bool merging,
                           float norm_factor,
                           int inp_size,
                           int total_count,
                           int num_seq,
                           int hidden,
                           int value_length)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int wid = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;
    int warp_num = blockDim.x >> 5;

    bool is_prompt = (value_length == num_seq);
    float2 query_value[8];
    float2* query_cast = reinterpret_cast<float2*>(query);
    float2* bias_cast = reinterpret_cast<float2*>(attn_bias);
    float2* key_cast = reinterpret_cast<float2*>(is_prompt ? query + (hidden << 1) : key_merged);
    float2* key_merged_cast;
    if (merging) key_merged_cast = reinterpret_cast<float2*>(key_merged);
    float2* new_key_cast = reinterpret_cast<float2*>(query + (hidden << 1));
    int input_offset = (blockIdx.x * warp_num + wid);

    int hidden31 = is_prompt ? hidden * 3 : hidden;
    if (input_offset < total_count) {
        query_cast += (input_offset % num_seq) * (hidden * 3) + (input_offset / num_seq) * inp_size;
        bias_cast += (input_offset / num_seq) * inp_size;
        int row = lane;
        int p = 0;

        while (row < inp_size) {
            query_value[p] = query_cast[row];
            if (attn_bias) {
                float2 bias_reg = bias_cast[row];
                query_value[p].x += bias_reg.x;
                query_value[p].y += bias_reg.y;
            } else {
                query_value[p].x *= norm_factor;
                query_value[p].y *= norm_factor;
            }

            p++;
            row += WARP_SIZE;
        }

        int seq_key = input_offset / num_seq;

        int unique_id = input_offset % num_seq;
        bias_cast += hidden;
        key_cast += (seq_key * inp_size);
        key_merged_cast += (seq_key * inp_size);

        int key_size = total_count / num_seq;
        int score_index = 0;

        if (seq_key < key_size) {
            {
                float scores[WARP_SIZE];
                int warp_iter;
                for (int i = 0; i < value_length; i += WARP_SIZE) {
                    warp_iter = (value_length - i) > WARP_SIZE ? WARP_SIZE : (value_length - i);
#pragma unroll
                    for (int p = 0; p < warp_iter; p++) { scores[p] = 0; }

                    for (int k = 0; k < warp_iter; k++) {
                        row = lane;
                        int p = 0;
                        while (row < inp_size) {
                            float2 key_value = key_cast[row];
                            if (attn_bias) {
                                float2 bias_reg = bias_cast[row];
                                key_value.x += bias_reg.x;
                                key_value.y += bias_reg.y;
                            }
                            key_value.x *= norm_factor;
                            key_value.y *= norm_factor;

                            if (is_prompt && (key_merged != nullptr) && unique_id == 0)
                                key_merged_cast[row] = key_value;

                            float2 mul;
                            mul.x = query_value[p].x * key_value.x;
                            mul.y = query_value[p].y * key_value.y;
                            scores[k] += (mul.x + mul.y);
                            row += WARP_SIZE;
                            p++;
                        }
                        key_cast += hidden31;
                        if (is_prompt && (key_merged != nullptr) && unique_id == 0)
                            key_merged_cast += (hidden);
#pragma unroll
                        for (int w = 1; w < WARP_SIZE; w *= 2)
                            scores[k] += g.shfl_xor(scores[k], w);
                    }
                    if (lane < (warp_iter >> 1)) {
                        shared_soft[wid * 1000 + ((lane + score_index) << 1)] = scores[(lane << 1)];
                        shared_soft[wid * 1000 + ((lane + score_index) << 1) + 1] =
                            scores[(lane << 1) + 1];
                    }
                    score_index += (warp_iter >> 1);
                }
                if (warp_iter % 2 == 1) {
                    if (lane == (warp_iter >> 1)) {
                        shared_soft[wid * 1000 + (score_index << 1)] = scores[(lane << 1)];
                    }
                }
            }
            if (!is_prompt && key_merged != nullptr) {
                new_key_cast += (((blockIdx.x * warp_num + wid) / num_seq) * inp_size);
                if (merging) {
                    key_merged_cast = reinterpret_cast<float2*>(key_merged);
                    key_merged_cast += ((blockIdx.x * warp_num + wid) / num_seq) * inp_size +
                                       ((hidden)*value_length);
                }
                row = lane;
                int p = 0;
                float score = 0;
                while (row < inp_size) {
                    float2 key_value = new_key_cast[row];
                    if (merging && unique_id == 0) key_merged_cast[row] = key_value;

                    key_value.x *= norm_factor;
                    key_value.y *= norm_factor;

                    float2 mul;
                    mul.x = query_value[p].x * key_value.x;
                    mul.y = query_value[p].y * key_value.y;
                    score += (mul.x + mul.y);
                    row += WARP_SIZE;
                    p++;
                }
#pragma unroll
                for (int w = 1; w < WARP_SIZE; w *= 2) score += g.shfl_down(score, w);

                if (lane == 0) {
                    if ((value_length + 1) % 2 == 0) {
                        shared_soft[wid * 1000 + (score_index << 1) + 1] = score;
                    } else {
                        shared_soft[wid * 1000 + (score_index << 1)] = score;
                    }
                }
            }
        }
    }
}

template <int tbSeq>
__device__ void attn_softmax(__half* shared_soft,
                             __half2* shared_soft1,
                             __half* mask,
                             int heads,
                             int total_count,
                             int num_seq,
                             int sequence_length,
                             bool triangular,
                             bool recompute)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    // int tbSeq = (sequence_length-1) / (WARP_SIZE << 2) + 1;
    int wid = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;
    int warp_num = blockDim.x >> 5;

    float2 low_data[tbSeq];
    float2 high_data[tbSeq];

    int iter_offset = blockIdx.x * (blockDim.x >> 5) + wid;
    if (iter_offset < total_count) {
        int iteration_stride = (blockDim.x >> 5) * gridDim.x;

        int seq_id = iter_offset % num_seq;
        int seq_id4 = seq_id >> 2;
        float max_val = minus_infinity;

        int mask_offset = (iter_offset / (heads * num_seq)) * (sequence_length);
        /**********
            [1 0 0 0]      [0 -inf -inf -inf]
            [1 1 0 0]      [0  0   -inf -inf]
            [1 1 1 0]      [0  0   0    -inf]
            [1 1 1 1]      [0  0   0       0]
        **********/
        for (int i = 0; i < tbSeq; i++) {
            int data_id = i * (WARP_SIZE << 2) + (lane << 2);
            if ((!triangular || ((data_id >> 2) <= seq_id4)) && data_id < sequence_length) {
                if ((sequence_length - data_id) >= 4) {
                    low_data[i].x = __half2float(shared_soft[wid * 1024 + data_id]);
                    low_data[i].y = (!triangular || ((data_id + 1) <= seq_id))
                                        ? __half2float(shared_soft[wid * 1024 + data_id + 1])
                                        : minus_infinity;
                    high_data[i].x = (!triangular || ((data_id + 2) <= seq_id))
                                         ? __half2float(shared_soft[wid * 1024 + data_id + 2])
                                         : minus_infinity;
                    high_data[i].y = (!triangular || ((data_id + 3) <= seq_id))
                                         ? __half2float(shared_soft[wid * 1024 + data_id + 3])
                                         : minus_infinity;
                    if (mask && !triangular && recompute) {
                        low_data[i].x += __half2float(mask[data_id + mask_offset]);
                        low_data[i].y += __half2float(mask[data_id + mask_offset + 1]);
                        high_data[i].x += __half2float(mask[data_id + mask_offset + 2]);
                        high_data[i].y += __half2float(mask[data_id + mask_offset + 3]);
                    }
                } else {
                    low_data[i].x = __half2float(shared_soft[wid * 1024 + data_id]);
                    low_data[i].y = (((!triangular || (data_id + 1) <= seq_id)) &&
                                     (data_id + 1) < sequence_length)
                                        ? __half2float(shared_soft[wid * 1024 + data_id + 1])
                                        : minus_infinity;
                    high_data[i].x = (((!triangular || (data_id + 2) <= seq_id)) &&
                                      (data_id + 2) < sequence_length)
                                         ? __half2float(shared_soft[wid * 1024 + data_id + 2])
                                         : minus_infinity;
                    high_data[i].y = minus_infinity;
                    if (mask && !triangular && recompute) {
                        low_data[i].x += __half2float(mask[data_id + mask_offset]);
                        if ((data_id + 1) < sequence_length)
                            low_data[i].y += __half2float(mask[data_id + mask_offset + 1]);
                        if ((data_id + 2) < sequence_length)
                            high_data[i].x += __half2float(mask[data_id + mask_offset + 2]);
                        // high_data[i].y += __half2float(mask[data_id + mask_offset + 3]);
                    }
                }
                max_val = (low_data[i].x > max_val ? low_data[i].x : max_val);
                max_val = (low_data[i].y > max_val ? low_data[i].y : max_val);
                max_val = (high_data[i].x > max_val ? high_data[i].x : max_val);
                max_val = (high_data[i].y > max_val ? high_data[i].y : max_val);
            } else {
                low_data[i].x = minus_infinity;
                low_data[i].y = minus_infinity;
                high_data[i].x = minus_infinity;
                high_data[i].y = minus_infinity;
            }
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) {
            auto temp = g.shfl_xor(max_val, i);
            max_val = (temp > max_val ? temp : max_val);
        }

        __shared__ float partialSum[MAX_WARP_NUM];

        if (Reduce_Threads > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = max_val;
            b.sync();

            if (lane < warp_num) max_val = partialSum[lane];

            int iters = warp_num;
            if (Reduce_Threads < iteration_stride) iters /= (iteration_stride / Reduce_Threads);

            for (int i = 1; i < iters; i *= 2) {
                auto temp = g.shfl_xor(max_val, i);
                max_val = (temp > max_val ? temp : max_val);
            }

            max_val = g.shfl(max_val, threadIdx.x / WARP_SIZE);
        }

        float sum = 0;
        for (int i = 0; i < tbSeq; i++) {
            low_data[i].x = __expf(low_data[i].x - max_val);
            low_data[i].y = __expf(low_data[i].y - max_val);
            high_data[i].x = __expf(high_data[i].x - max_val);
            high_data[i].y = __expf(high_data[i].y - max_val);

            sum += (low_data[i].x + low_data[i].y + high_data[i].x + high_data[i].y);
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) sum += g.shfl_xor(sum, i);

        if (Reduce_Threads > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = sum;
            b.sync();

            if (lane < warp_num) sum = partialSum[lane];

            int iters = warp_num;
            if (Reduce_Threads < iteration_stride) iters /= (iteration_stride / Reduce_Threads);

            for (int i = 1; i < iters; i *= 2) { sum += g.shfl_xor(sum, i); }

            sum = g.shfl(max_val, threadIdx.x / WARP_SIZE);
        }
        sum += 1e-6;
        for (int i = 0; i < tbSeq; i++) {
            int data_id = i * (WARP_SIZE << 2) + (lane << 2);

            if (data_id < sequence_length) {
                if ((sequence_length - data_id) >= 4) {
                    low_data[i].x /= sum;
                    low_data[i].y /= sum;
                    high_data[i].x /= sum;
                    high_data[i].y /= sum;
                } else {
                    low_data[i].x /= sum;
                    low_data[i].y = (((data_id + 1) < sequence_length) ? low_data[i].y / sum : 0.f);
                    high_data[i].x =
                        (((data_id + 2) < sequence_length) ? high_data[i].x / sum : 0.f);
                    high_data[i].y = 0;
                }
                shared_soft1[wid * 1024 + (data_id >> 1)] = __float22half2_rn(low_data[i]);
                shared_soft1[wid * 1024 + (data_id >> 1) + 1] = __float22half2_rn(high_data[i]);
            }
        }
    }
}

template <int tbSeq>
__device__ void attn_softmax(float* shared_soft,
                             float2* shared_soft1,
                             float* mask,
                             int heads,
                             int total_count,
                             int num_seq,
                             int sequence_length,
                             bool triangular,
                             bool recompute)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    constexpr int reg_size = tbSeq << 1;

    int wid = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;
    int warp_num = blockDim.x >> 5;

    float2 val_data[reg_size];

    int iter_offset = blockIdx.x * (blockDim.x >> 5) + wid;
    if (iter_offset < total_count) {
        int iteration_stride = (blockDim.x >> 5) * gridDim.x;

        int seq_id = iter_offset % num_seq;
        int seq_id4 = seq_id >> 1;
        float max_val = minus_infinity;

        int mask_offset = (iter_offset / (heads * num_seq)) * (sequence_length);
        /**********
            [1 0 0 0]      [0 -inf -inf -inf]
            [1 1 0 0]      [0  0   -inf -inf]
            [1 1 1 0]      [0  0   0    -inf]
            [1 1 1 1]      [0  0   0       0]
        **********/

        for (int i = 0; i < reg_size; i++) {
            int data_id = i * (WARP_SIZE << 1) + (lane << 1);
            if ((!triangular || ((data_id >> 1) <= seq_id4)) && data_id < sequence_length) {
                if ((sequence_length - data_id) >= 2) {
                    val_data[i].x = shared_soft[wid * 1000 + data_id];
                    val_data[i].y = (!triangular || ((data_id + 1) <= seq_id))
                                        ? shared_soft[wid * 1000 + data_id + 1]
                                        : minus_infinity;

                    if (mask && !triangular && recompute) {
                        val_data[i].x += mask[data_id + mask_offset];
                        val_data[i].y += mask[data_id + mask_offset + 1];
                    }
                } else {
                    val_data[i].x = shared_soft[wid * 1000 + data_id];
                    val_data[i].y = minus_infinity;

                    if (mask && !triangular && recompute) {
                        val_data[i].x += mask[data_id + mask_offset];
                    }
                }
                max_val = (val_data[i].x > max_val ? val_data[i].x : max_val);
                max_val = (val_data[i].y > max_val ? val_data[i].y : max_val);
            } else {
                val_data[i].x = minus_infinity;
                val_data[i].y = minus_infinity;
            }
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) {
            auto temp = g.shfl_xor(max_val, i);
            max_val = (temp > max_val ? temp : max_val);
        }

        __shared__ float partialSum[MAX_WARP_NUM];

        if (Reduce_Threads > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = max_val;
            b.sync();

            if (lane < warp_num) max_val = partialSum[lane];

            int iters = warp_num;
            if (Reduce_Threads < iteration_stride) iters /= (iteration_stride / Reduce_Threads);

            for (int i = 1; i < iters; i *= 2) {
                auto temp = g.shfl_xor(max_val, i);
                max_val = (temp > max_val ? temp : max_val);
            }

            max_val = g.shfl(max_val, threadIdx.x / WARP_SIZE);
        }

        float sum = 0;
        for (int i = 0; i < reg_size; i++) {
            val_data[i].x = __expf(val_data[i].x - max_val);
            val_data[i].y = __expf(val_data[i].y - max_val);

            sum += (val_data[i].x + val_data[i].y);
        }

        for (int i = 1; i < WARP_SIZE; i *= 2) sum += g.shfl_xor(sum, i);

        if (Reduce_Threads > WARP_SIZE) {
            if (lane == 0) partialSum[wid] = sum;
            b.sync();

            if (lane < warp_num) sum = partialSum[lane];

            int iters = warp_num;
            if (Reduce_Threads < iteration_stride) iters /= (iteration_stride / Reduce_Threads);

            for (int i = 1; i < iters; i *= 2) { sum += g.shfl_xor(sum, i); }

            sum = g.shfl(max_val, threadIdx.x / WARP_SIZE);
        }
        sum += 1e-6;
        for (int i = 0; i < reg_size; i++) {
            int data_id = i * (WARP_SIZE << 1) + (lane << 1);

            if (data_id < sequence_length) {
                if ((sequence_length - data_id) >= 2) {
                    val_data[i].x /= sum;
                    val_data[i].y /= sum;
                } else {
                    val_data[i].x /= sum;
                    val_data[i].y = 0;
                }
                shared_soft1[wid * 1000 + (data_id >> 1)] = val_data[i];
            }
        }
    }
}

__device__ void attn_context(__half2* shared_soft1,
                             __half* prev_value,
                             __half* merged_value,
                             __half* attn_bias,
                             bool merging,
                             __half* output,
                             int value_length,
                             int num_seq,
                             int hidden,
                             int head_size,
                             int total_count)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    __half2* output_cast = reinterpret_cast<__half2*>(output);

    int wid = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;
    int warp_num = blockDim.x >> 5;
    bool is_prompt = (value_length == num_seq);
    __half2* value_cast =
        reinterpret_cast<__half2*>(is_prompt ? prev_value + 2 * (hidden << 1) : merged_value);
    __half2* new_value_cast = reinterpret_cast<__half2*>(prev_value + 2 * (hidden << 1));
    __half2* merged_value_cast = reinterpret_cast<__half2*>(merged_value);
    __half2* bias_cast;
    if (attn_bias) bias_cast = reinterpret_cast<__half2*>(attn_bias + 2 * (hidden << 1));
    int hidden31 = is_prompt ? (hidden)*3 : (hidden);

    int col_id = (blockIdx.x * warp_num + wid);
    int offset = col_id / num_seq;
    int value_size = total_count / num_seq;

    if (offset < value_size) {
        int wid_iter = 0;
        float2 sum[attn_warps << 1];
#pragma unroll
        for (int p = 0; p < (attn_warps << 1); p++) {
            sum[p].x = 0;
            sum[p].y = 0;
        }
        offset = (offset * head_size);

        int merge_offset = offset + lane;
        while (wid_iter < value_length) {
            __half2 val_h[2];
            __half* inp_data[2];

            val_h[0] = shared_soft1[wid * 1024 + (wid_iter >> 1)];
            val_h[1] = shared_soft1[wid * 1024 + (wid_iter >> 1) + 1];

            inp_data[0] = reinterpret_cast<__half*>(&val_h[0]);
            inp_data[1] = reinterpret_cast<__half*>(&val_h[1]);

            int row = lane;
            int iter = 0;
            int offset1 = offset + lane;

            if (merged_value != nullptr) merged_value_cast += merge_offset;
            while (row < head_size) {
                __half2 weight_h[4];
#pragma unroll
                for (int f = 0; f < 4; f++)
                    weight_h[f] = (wid_iter + f) < value_length ? value_cast[f * hidden31 + offset1]
                                                                : __float2half2_rn(0.f);

                if ((col_id % num_seq) == 0 && (merged_value != nullptr)) {
#pragma unroll
                    for (int f = 0; f < 4; f++)
                        if ((wid_iter + f) < value_length)
                            merged_value_cast[f * hidden] = weight_h[f];
                }
                if (attn_bias) {
                    __half2 bias_reg = bias_cast[offset1 % hidden];
#pragma unroll
                    for (int f = 0; f < 4; f++) {
                        weight_h[f].x += bias_reg.x;
                        weight_h[f].y += bias_reg.y;
                    }
                }
                {
                    float2 mul[4];
                    mul[0] = __half22float2(weight_h[0] *
                                            __halves2half2(inp_data[0][0], inp_data[0][0]));
                    mul[1] = __half22float2(weight_h[1] *
                                            __halves2half2(inp_data[0][1], inp_data[0][1]));
                    mul[2] = __half22float2(weight_h[2] *
                                            __halves2half2(inp_data[1][0], inp_data[1][0]));
                    mul[3] = __half22float2(weight_h[3] *
                                            __halves2half2(inp_data[1][1], inp_data[1][1]));

                    sum[iter].x += mul[0].x + mul[1].x + mul[2].x + mul[3].x;
                    sum[iter].y += mul[0].y + mul[1].y + mul[2].y + mul[3].y;
                }
                row += (WARP_SIZE);
                offset1 += (WARP_SIZE);
                if (merged_value != nullptr) merged_value_cast += WARP_SIZE;
                iter++;
            }
            if (merged_value != nullptr)
                merged_value_cast = reinterpret_cast<__half2*>(merged_value);
            wid_iter += 4;
            offset += (hidden31 << 2);
            merge_offset += (hidden << 2);
        }

        if (!is_prompt && (merged_value != nullptr)) {
            int row = lane;
            int merge_offset = (col_id / num_seq) * head_size + lane + (value_length * hidden);
            __half2 val_h;
            val_h = shared_soft1[wid * 1024 + (value_length >> 1)];
            __half* inp_data;
            inp_data = reinterpret_cast<__half*>(&val_h);
            __half2 vals_f = __halves2half2(inp_data[value_length % 2], inp_data[value_length % 2]);
            int p = 0;
            int offset1 = (col_id / num_seq) * (head_size) + lane;
            while (row < head_size) {
                __half2 new_value_data = new_value_cast[offset1];
                float2 mul = __half22float2(vals_f * new_value_data);
                sum[p].x += mul.x;
                sum[p].y += mul.y;
                if ((col_id % num_seq) == 0) merged_value_cast[merge_offset] = new_value_data;
                row += WARP_SIZE;
                offset1 += WARP_SIZE;
                merge_offset += WARP_SIZE;
                p++;
            }
        }
        if (col_id < total_count) {
            int p = 0;
            int row = lane;
            col_id = col_id * head_size + lane;
            while (row < head_size) {
                output_cast[col_id] = __float22half2_rn(sum[p]);
                row += WARP_SIZE;
                col_id += WARP_SIZE;
                p++;
            }
        }
    }
}

__device__ void attn_context(float2* shared_soft1,
                             float* prev_value,
                             float* merged_value,
                             float* attn_bias,
                             bool merging,
                             float* output,
                             int value_length,
                             int num_seq,
                             int hidden,
                             int head_size,
                             int total_count)
{
    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    float2* output_cast = reinterpret_cast<float2*>(output);

    float2 ZERO_f2;
    ZERO_f2.x = ZERO_f2.y = 0.f;

    int wid = threadIdx.x >> 5;
    int lane = threadIdx.x & 0x1f;
    int warp_num = blockDim.x >> 5;
    bool is_prompt = (num_seq == value_length);
    float2* bias_cast;
    if (attn_bias) bias_cast = reinterpret_cast<float2*>(attn_bias + 2 * (hidden << 1));
    float2* value_cast =
        reinterpret_cast<float2*>(is_prompt ? prev_value + 2 * (hidden << 1) : merged_value);
    float2* new_value_cast = reinterpret_cast<float2*>(prev_value + 2 * (hidden << 1));
    float2* merged_value_cast;
    if (merging) merged_value_cast = reinterpret_cast<float2*>(merged_value);
    int hidden31 = is_prompt ? (hidden)*3 : (hidden);
    int offset = (blockIdx.x * warp_num + wid) / num_seq;
    int value_size = total_count / num_seq;
    int unique_id = (blockIdx.x * warp_num + wid) % num_seq;
    if (offset < value_size) {
        float2 val_data;
        int wid_iter = 0;
        float2 sum[8];
#pragma unroll
        for (int p = 0; p < 8; p++) {
            sum[p].x = 0;
            sum[p].y = 0;
        }
        offset = (offset * head_size);
        int merge_offset = offset + lane;
        while (wid_iter < value_length) {
            {
                val_data = shared_soft1[wid * 1000 + (wid_iter >> 1)];
            }
            int row = lane;
            int offset1 = lane + offset;
            int merge_offset1 = merge_offset;
            int iter = 0;
            while (row < head_size) {
                float2 weight[2];
                weight[0] = value_cast[offset1];
                weight[1] =
                    ((wid_iter + 1) < value_length ? value_cast[hidden31 + offset1] : ZERO_f2);
                if ((merged_value != nullptr) && unique_id == 0) {
                    merged_value_cast[merge_offset1] = weight[0];
                    if ((wid_iter + 1) < value_length)
                        merged_value_cast[hidden + merge_offset1] = weight[1];
                }
                if (attn_bias) {
                    float2 bias_reg = bias_cast[offset1 % hidden];
                    weight[0].x += bias_reg.x;
                    weight[0].y += bias_reg.y;
                    weight[1].x += bias_reg.x;
                    weight[1].y += bias_reg.y;
                }
                float2 mul[2];
                {
                    mul[0].x = val_data.x * weight[0].x;
                    mul[0].y = val_data.x * weight[0].y;
                    mul[1].x = val_data.y * weight[1].x;
                    mul[1].y = val_data.y * weight[1].y;

                    sum[iter].x += mul[0].x + mul[1].x;
                    sum[iter].y += mul[0].y + mul[1].y;
                }
                row += (WARP_SIZE);
                offset1 += (WARP_SIZE);
                merge_offset1 += WARP_SIZE;
                iter++;
            }
            wid_iter += 2;
            offset += (hidden31 * 2);
            merge_offset += (hidden * 2);
        }

        if (!is_prompt && (merged_value != nullptr)) {
            int row = lane;
            int merge_offset = ((blockIdx.x * warp_num + wid) / num_seq) * head_size + lane +
                               (value_length * hidden);
            val_data = shared_soft1[wid * 1000 + (value_length >> 1)];

            int p = 0;
            int offset1 = ((blockIdx.x * warp_num + wid) / num_seq) * (head_size) + lane;
            while (row < head_size) {
                float2 new_value_data = new_value_cast[offset1];
                float2 mul;

                mul.x = ((value_length % 2) ? val_data.y : val_data.x) * new_value_data.x;
                mul.y = ((value_length % 2) ? val_data.y : val_data.x) * new_value_data.y;

                sum[p].x += mul.x;
                sum[p].y += mul.y;
                if (merging && unique_id == 0) merged_value_cast[merge_offset] = new_value_data;
                row += WARP_SIZE;
                offset1 += WARP_SIZE;
                merge_offset += WARP_SIZE;
                p++;
            }
        }
        int offset1 = ((blockIdx.x * warp_num + wid));
        if (offset1 < total_count) {
            int p = 0;
            int row = lane;
            offset1 = offset1 * head_size + lane;
            while (row < head_size) {
                output_cast[offset1] = sum[p];
                row += WARP_SIZE;
                offset1 += WARP_SIZE;
                p++;
            }
        }
    }
}

template <int tbSize, int tbSeq>
__global__ void attn_softmax_context(__half* output,
                                     __half* query,
                                     __half* mask,
                                     float norm_factor,
                                     __half* key_merged,
                                     __half* merged_value,
                                     __half* attn_bias,
                                     bool merging,
                                     bool triangular,
                                     bool recompute,
                                     int total_count,
                                     int heads,
                                     int head_size,
                                     int value_length,
                                     int seq_length,
                                     int num_seq,
                                     float scale)
{
#if __CUDA_ARCH__ >= 700

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int iter_offset = blockIdx.x * (blockDim.x >> 5) + (threadIdx.x >> 5);

    if (iter_offset < total_count) {
        __shared__ __half2 shared_soft1[attn_warps * (1024 + 1)];
        int hidden = heads * head_size;
        {
            __shared__ __half shared_soft[attn_warps * (1024 + 1)];
            // Attntion_Score
            attn_score(shared_soft,
                       query,
                       key_merged,
                       attn_bias,
                       merging,
                       norm_factor,
                       (head_size >> 1),
                       total_count,
                       num_seq,
                       hidden,
                       value_length);
            b.sync();
            attn_softmax<tbSeq>(shared_soft,
                                shared_soft1,
                                mask,
                                heads,
                                total_count,
                                num_seq,
                                seq_length,
                                triangular,
                                recompute);
            b.sync();
        }
        // Attention_Context
        attn_context(shared_soft1,
                     query,  // prev_value,
                     merged_value,
                     attn_bias,
                     merging,
                     output,
                     value_length,
                     num_seq,
                     hidden,
                     head_size,
                     total_count);
    }
#endif
}

template <int tbSize, int tbSeq>
__global__ void attn_softmax_context(float* output,
                                     float* query,
                                     float* mask,
                                     float norm_factor,
                                     float* key_merged,
                                     float* merged_value,
                                     float* attn_bias,
                                     bool merging,
                                     bool triangular,
                                     bool recompute,
                                     int total_count,
                                     int heads,
                                     int head_size,
                                     int value_length,
                                     int seq_length,
                                     int num_seq,
                                     float scale)
{
#if __CUDA_ARCH__ >= 700

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int iter_offset = blockIdx.x * (blockDim.x >> 5) + (threadIdx.x >> 5);

    // if (iter_offset < total_count)
    {
        __shared__ float2 shared_soft1[4004];
        int hidden = heads * head_size;
        {
            __shared__ float shared_soft[4004];
            // Attntion_Score
            attn_score(shared_soft,
                       query,
                       key_merged,
                       attn_bias,
                       merging,
                       norm_factor,
                       head_size,
                       total_count,
                       num_seq,
                       hidden,
                       value_length);
            b.sync();
            attn_softmax<tbSeq>(shared_soft,
                                shared_soft1,
                                mask,
                                heads,
                                total_count,
                                num_seq,
                                seq_length,
                                triangular,
                                recompute);
            // return;
            b.sync();
        }
        // Attention_Context
        attn_context(shared_soft1,
                     query,
                     merged_value,
                     attn_bias,
                     merging,
                     output,
                     value_length,
                     num_seq,
                     hidden,
                     head_size,
                     total_count);
    }
#endif
}

template <typename T>
void launch_attn_softmax_context(T* out,
                                 T* query,
                                 T* mask,
                                 float norm_factor,
                                 T* key_merged,
                                 T* merged_value,
                                 T* attn_bias,
                                 bool merging,
                                 bool triangular,
                                 bool recompute,
                                 int batch_size,
                                 int heads,
                                 int head_size,
                                 int value_length,
                                 int num_seq,
                                 int sequence_length,
                                 float scale,
                                 hipStream_t stream)
{
    int total_count = batch_size * heads * num_seq;

    dim3 grid_dim((total_count - 1) / attn_warps + 1);
    dim3 block_dim(Attn_Threads_111);
    if (sequence_length <= 128)
        attn_softmax_context<32, 1><<<grid_dim, block_dim, 0, stream>>>(out,
                                                                        query,
                                                                        mask,
                                                                        norm_factor,
                                                                        key_merged,
                                                                        merged_value,
                                                                        attn_bias,
                                                                        merging,
                                                                        triangular,
                                                                        recompute,
                                                                        total_count,
                                                                        heads,
                                                                        head_size / 2,
                                                                        value_length,
                                                                        sequence_length,
                                                                        num_seq,
                                                                        scale);
    else if (sequence_length <= 256)
        attn_softmax_context<32, 2><<<grid_dim, block_dim, 0, stream>>>(out,
                                                                        query,
                                                                        mask,
                                                                        norm_factor,
                                                                        key_merged,
                                                                        merged_value,
                                                                        attn_bias,
                                                                        merging,
                                                                        triangular,
                                                                        recompute,
                                                                        total_count,
                                                                        heads,
                                                                        head_size / 2,
                                                                        value_length,
                                                                        sequence_length,
                                                                        num_seq,
                                                                        scale);
    else if (sequence_length <= 512)
        attn_softmax_context<32, 4><<<grid_dim, block_dim, 0, stream>>>(out,
                                                                        query,
                                                                        mask,
                                                                        norm_factor,
                                                                        key_merged,
                                                                        merged_value,
                                                                        attn_bias,
                                                                        merging,
                                                                        triangular,
                                                                        recompute,
                                                                        total_count,
                                                                        heads,
                                                                        head_size / 2,
                                                                        value_length,
                                                                        sequence_length,
                                                                        num_seq,
                                                                        scale);
    else if (sequence_length <= 1024)
        attn_softmax_context<32, 8><<<grid_dim, block_dim, 0, stream>>>(out,
                                                                        query,
                                                                        mask,
                                                                        norm_factor,
                                                                        key_merged,
                                                                        merged_value,
                                                                        attn_bias,
                                                                        merging,
                                                                        triangular,
                                                                        recompute,
                                                                        total_count,
                                                                        heads,
                                                                        head_size / 2,
                                                                        value_length,
                                                                        sequence_length,
                                                                        num_seq,
                                                                        scale);
    else if (sequence_length <= 2048)
        attn_softmax_context<32, 16><<<grid_dim, block_dim, 0, stream>>>(out,
                                                                         query,
                                                                         mask,
                                                                         norm_factor,
                                                                         key_merged,
                                                                         merged_value,
                                                                         attn_bias,
                                                                         merging,
                                                                         triangular,
                                                                         recompute,
                                                                         total_count,
                                                                         heads,
                                                                         head_size / 2,
                                                                         value_length,
                                                                         sequence_length,
                                                                         num_seq,
                                                                         scale);
    else if (sequence_length <= 4096)
        attn_softmax_context<32, 32><<<grid_dim, block_dim, 0, stream>>>(out,
                                                                         query,
                                                                         mask,
                                                                         norm_factor,
                                                                         key_merged,
                                                                         merged_value,
                                                                         attn_bias,
                                                                         merging,
                                                                         triangular,
                                                                         recompute,
                                                                         total_count,
                                                                         heads,
                                                                         head_size / 2,
                                                                         value_length,
                                                                         sequence_length,
                                                                         num_seq,
                                                                         scale);
    else
        throw std::runtime_error(
            "Unsupport Seq_Length! Check the restriction of the max_threads and "
            "max_thread_iterations!");
}

template void launch_attn_softmax_context(float* out,
                                          float* query,
                                          float* mask,
                                          float norm_factor,
                                          float* key_merged,
                                          float* merged_value,
                                          float* attn_bias,
                                          bool merging,
                                          bool triangular,
                                          bool recompute,
                                          int batch_size,
                                          int heads,
                                          int head_size,
                                          int value_length,
                                          int num_seq,
                                          int sequence_length,
                                          float scale,
                                          hipStream_t stream);

template void launch_attn_softmax_context(__half* out,
                                          __half* query,
                                          __half* mask,
                                          float norm_factor,
                                          __half* key_merged,
                                          __half* merged_value,
                                          __half* attn_bias,
                                          bool merging,
                                          bool triangular,
                                          bool recompute,
                                          int batch_size,
                                          int heads,
                                          int head_size,
                                          int value_length,
                                          int num_seq,
                                          int sequence_length,
                                          float scale,
                                          hipStream_t stream);
