#include "hip/hip_runtime.h"
#include "custom_cuda_layers.h"
#include <hip/hip_runtime_api.h>
namespace cg = cooperative_groups;

__global__ void transform_scale(float* query, float* kv_cache,
                                const float* vals,
                                int hidden_dim,
                                int seq_length,
                                unsigned cur_tokens,
                                uint64_t value_offset,
                                int heads,
                                int head_ext,
                                float norm_factor)
{
    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0_out_stride = d0_stride;
    int d1_out_stride = d2_stride;
    int d2_out_stride = d2_stride * seq_length;

    int d0 = blockIdx.x;                                                  // Batch
    int d1 = blockIdx.y;                                                  // Sequence ID (0-127)
    int cnt = blockIdx.z / head_ext;                                      // Hidden count
    int d2 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head (0-11)
    int d3 = threadIdx.x;                                                 // Values (groups of 4)
    kv_cache += d0 * (hidden_dim * cur_tokens);
    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    float4* output_vec = reinterpret_cast<float4*>(cnt == 0 ? query : (cnt == 1 ? kv_cache : kv_cache + value_offset));

    float4 inputs = vals_vec[d0 * d0_stride * (gridDim.z / head_ext) + cnt * d1_stride +
                             d1 * d1_stride * (gridDim.z / head_ext) + d2 * d2_stride + d3];

    inputs.x = (cnt < 2) ? inputs.x  * norm_factor : inputs.x;
    inputs.y = (cnt < 2) ? inputs.y  * norm_factor : inputs.y;
    inputs.z = (cnt < 2) ? inputs.z  * norm_factor : inputs.z;
    inputs.w = (cnt < 2) ? inputs.w  * norm_factor : inputs.w;
    
    output_vec[d0 * d0_out_stride + d1 * d1_out_stride +
               d2 * d2_out_stride + d3] = inputs;
}

__global__ void transform_scale(__half* query, __half* kv_cache,
                                const __half* vals,
                                unsigned hidden_dim,
                                int seq_length,
                                unsigned cur_tokens,
                                uint64_t value_offset,
                                int heads,
                                int head_ext,
                                float norm_factor)
{


    //int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    //int d0 = blockIdx.x;                                                  // Batch
    int d1 = blockIdx.y;                                                  // Sequence ID (0-127)
    int cnt = blockIdx.z / head_ext;                                      // Hidden count
    int d2 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head (0-11)
    int d3 = threadIdx.x;                                                 // Values (groups of 4)

    int d2_out_stride = d2_stride * (cnt == 0 ? seq_length : MAX_OUT_TOKES);

    

    float4 vals_arr;
    float4 bias_arr;
    float4 output_arr;
    __half2* vals_half = reinterpret_cast<__half2*>(&vals_arr);
    //__half2* bias_half = reinterpret_cast<__half2*>(&bias_arr);
    __half2* output_half = reinterpret_cast<__half2*>(&output_arr);

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    //const float4* bias_vec = reinterpret_cast<const float4*>(bias);
    float4* output_vec = reinterpret_cast<float4*>(cnt == 0 ? query : (cnt == 1 ? kv_cache : kv_cache + value_offset));

    //vals_vec += (d0 * d0_stride * (gridDim.z / head_ext));
    vals_vec += (d1 * d1_stride * (gridDim.z / head_ext));
    vals_vec += (cnt * d1_stride);
    vals_vec += (d2 * d2_stride);

    //bias_vec += (cnt * d1_stride);
    //bias_vec += (d2 * d2_stride);

    //output_vec += (d0_stride * gridDim.x);
    output_vec += (d1 * d2_stride);
    //output_vec += (d0 * d0_stride);
    output_vec += (d2 * d2_out_stride);

    //bias_arr = bias_vec[d3];
    vals_arr = vals_vec[d3];

    output_half[0] = vals_half[0] ;//+ bias_half[0];
    output_half[1] = vals_half[1] ;//+ bias_half[1];
    output_half[2] = vals_half[2] ;//+ bias_half[2];
    output_half[3] = vals_half[3] ;//+ bias_half[3];
    output_vec[d3] = output_arr;
/*
    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;


    int d0 = blockIdx.x;                                                  // Batch
    int d1 = blockIdx.y;                                                  // Sequence ID (0-127)
    int cnt = blockIdx.z / head_ext;                                      // Hidden count
    int d2 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head (0-11)
    int d3 = threadIdx.x;                                                 // Values (groups of 4)

    int d2_out_stride = d2_stride * (cnt == 0 ? seq_length : MAX_OUT_TOKES);
    float4 vals_arr;
    float4 output_arr;
    __half2* vals_half = reinterpret_cast<__half2*>(&vals_arr);
    __half2* output_half = reinterpret_cast<__half2*>(&output_arr);
    __half2 norm_factor_h = __float2half2_rn(norm_factor);
    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    float4* output_vec = reinterpret_cast<float4*>(cnt == 0 ? query : (cnt == 1 ? kv_cache : (kv_cache + value_offset)));

    if (cnt > 0) output_vec += d0 * (hidden_dim * cur_tokens);

    vals_vec += (d0 * d0_stride * (gridDim.z / head_ext));
    vals_vec += (d1 * d1_stride * (gridDim.z / head_ext));
    vals_vec += (cnt * d1_stride);
    vals_vec += (d2 * d2_stride);

    output_vec += (d1 * d2_stride);
    output_vec += (d0 * d0_stride);
    output_vec += (d2 * d2_out_stride);

    vals_arr = vals_vec[d3];

    output_half[0] = (cnt < 2) ? vals_half[0] * norm_factor_h : vals_half[0];
    output_half[1] = (cnt < 2) ? vals_half[1] * norm_factor_h : vals_half[1];
    output_half[2] = (cnt < 2) ? vals_half[2] * norm_factor_h : vals_half[2];
    output_half[3] = (cnt < 2) ? vals_half[3] * norm_factor_h : vals_half[3];
        
    output_vec[d3] = output_arr;
*/

}

// [B S C*H] - > C * [B A S N]
template <>
void launch_transform_scale<float>(float* vals,
                                   float* query,
                                   float* kv_cache,
                                   int batch_size,
                                   int seq_length,
                                   unsigned cur_tokens,
                                   size_t value_offset,
                                   unsigned hidden_dim,
                                   int heads,
                                   hipStream_t stream,
                                   int trans_count,
                                   float norm_factor)
{
    hidden_dim >>= 2;
    int head_ext = (hidden_dim - 1) / MAX_THREADS + 1;

    dim3 block_dim(hidden_dim / heads, (heads / head_ext));
    dim3 grid_dim(batch_size, seq_length, (trans_count * head_ext));

    transform_scale<<<grid_dim, block_dim, 0, stream>>>(
        query, kv_cache, vals, hidden_dim, seq_length, cur_tokens, value_offset, heads, head_ext, norm_factor);
}

template <>
void launch_transform_scale<__half>(__half* vals,
                                    __half* query,
                                    __half* kv_cache,
                                    int batch_size,
                                    int seq_length,
                                    unsigned cur_tokens,
                                    size_t value_offset,
                                    unsigned hidden_dim,
                                    int heads,
                                    hipStream_t stream,
                                    int trans_count,
                                    float norm_factor)
{
    hidden_dim >>= 3;
    int head_ext = (hidden_dim - 1) / MAX_THREADS + 1;
    dim3 block_dim(hidden_dim / heads, (heads / head_ext));
    dim3 grid_dim(batch_size, seq_length, (trans_count * head_ext));
    transform_scale<<<grid_dim, block_dim, 0, stream>>>(
        query, kv_cache, vals, hidden_dim, seq_length, cur_tokens, value_offset,  heads, head_ext, norm_factor);
}

// Bias add

__global__ void bias_add_transform_0213(float* output,
                                               const float* vals,
                                               const float* bias,
                                               int hidden_dim,
                                               int seq_length,
                                               int heads,
                                               int head_ext)
{
    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0_out_stride = d0_stride;
    int d1_out_stride = d2_stride;
    int d2_out_stride = d2_stride * seq_length;

    int d0 = blockIdx.x;                                                  // Batch
    int d1 = blockIdx.y;                                                  // Sequence ID (0-127)
    int cnt = blockIdx.z / head_ext;                                      // Hidden count
    int d2 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head (0-11)
    int d3 = threadIdx.x;                                                 // Values (groups of 4)

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    const float4* bias_vec = reinterpret_cast<const float4*>(bias);
    float4* output_vec = reinterpret_cast<float4*>(output);

    float4 inputs = vals_vec[d0 * d0_stride * (gridDim.z / head_ext) + cnt * d1_stride +
                             d1 * d1_stride * (gridDim.z / head_ext) + d2 * d2_stride + d3];
    float4 biases = bias_vec[cnt * d1_stride + d2 * d2_stride + d3];

    float4 outputs;
    outputs.x = inputs.x + biases.x;
    outputs.y = inputs.y + biases.y;
    outputs.z = inputs.z + biases.z;
    outputs.w = inputs.w + biases.w;

    output_vec[cnt * d0_out_stride * gridDim.x + d0 * d0_out_stride + d1 * d1_out_stride +
               d2 * d2_out_stride + d3] = outputs;
}

#define ATTN_H 3
#define MAX_SEQ_LINE 10

__global__ void bias_add_transform_0213(__half* output, //q
                                                __half* k_cache,
                                                __half* v_cache,
                                                const __half* vals, //qkv
                                                const __half* bias,
                                                int hidden_dim,
                                                int seq_length,
                                                unsigned seq_offset,
                                                int all_tokens,
                                                int heads,
                                                int rotary_dim,
                                                bool rotate_half,
                                                bool rotate_every_two,
                                                int head_ext)
{
#if __CUDA_ARCH__ >= 700

    unsigned half_dim = (rotary_dim << 3) >> 1;
    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0 = blockIdx.x;                                                  // Batch
    int d1 = blockIdx.y;                                                   // Sequence ID (0-127)
    int cnt = blockIdx.z;                                                  // Hidden count
    int d2 = threadIdx.y;                                                  // Head (0-11)
    int d3 = threadIdx.x;                                                 // Values (groups of 4)


    int d2_out_stride = d2_stride * (cnt == 0 ? seq_length : MAX_OUT_TOKES);
    float4 vals_arr;
    float4 output_arr;
    
    __half2* vals_half = reinterpret_cast<__half2*>(&vals_arr);
    __half2* output_half = reinterpret_cast<__half2*>(&output_arr);

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    float4* output_vec = reinterpret_cast<float4*>(cnt == 0 ? output : (cnt == 1 ? k_cache : v_cache));

    vals_vec += (d0 * d0_stride * (gridDim.z / head_ext));
    vals_vec += (d1 * d1_stride * (gridDim.z / head_ext));
    vals_vec += (cnt * d1_stride);
    vals_vec += (d2 * d2_stride);

    output_vec += (d1 * d2_stride);
    output_vec += (d0 * d0_stride);
    output_vec += (d2 * d2_out_stride);

    unsigned seq_id = d1 + seq_offset;

    int lane = d3 & 0x1f;
    if (cnt < 2 && rotary_dim > 0 && d3 < rotary_dim) {
        float4 q = vals_vec[d3]; 
        __half2* q_h = reinterpret_cast<__half2*>(&q);
        if (rotate_every_two){
            #pragma unroll
            for (int o = 0;o < 4;o++){
                float inv_freq = (float)(((d3 << 2) + o) * 2) / (float)(rotary_dim << 3);
                inv_freq = 1.0 / powf(10000.0, inv_freq) * (float)seq_id;
                float q_data[2];
                q_data[0] = (float)q_h[o].x;
                q_data[1] = (float)q_h[o].y;
                q_h[o].x = (__half)(-1.0 * q_data[1] * sinf(inv_freq) + q_data[0] * cosf(inv_freq));
                q_h[o].y = (__half)(q_data[0] * sinf(inv_freq) + q_data[1] * cosf(inv_freq));
            }
        }
        //else{
        //    float4 q = vals_vec[d3]; 
        //    float4 q_rot;
        //    __half2* q_h = reinterpret_cast<__half2*>(&q);
        //    {
        //        __half2* qrot_h = reinterpret_cast<__half2*>(&q_rot);
        //        #pragma unroll
        //        for (int o = 0;o < 4;o++){
        //            int index = ((d3 << 2) + o) * 2;
        //            float rotary_sign = (index > (half_dim - 1) ? -1.0 : 1.0);
        //            qrot_h[o].x = __float2half((float)q_h[o].x * rotary_sign);
        //            qrot_h[o].y = __float2half((float)q_h[o].y * rotary_sign);
        //        }
        //    }
        //    __half2* qrot_h = reinterpret_cast<__half2*>(&q_rot);
        //    {
        //        float4 q_rot_tmp;
        //        float4 q_rot_tmp1;
        //        {
        //            q_rot_tmp.x = __shfl_sync(0xffffffff, q_rot.x, 1);
        //            q_rot_tmp.y = __shfl_sync(0xffffffff, q_rot.y, 1);
        //            q_rot_tmp.z = __shfl_sync(0xffffffff, q_rot.z, 1);
        //            q_rot_tmp.w = __shfl_sync(0xffffffff, q_rot.w, 1);
        //        }
        //        
        //        if(lane < 2){
        //            q_rot_tmp.x = __shfl_xor_sync(0xffffffff, q_rot.x, 1);
        //            q_rot_tmp.y = __shfl_xor_sync(0xffffffff, q_rot.y, 1);
        //            q_rot_tmp.z = __shfl_xor_sync(0xffffffff, q_rot.z, 1);
        //            q_rot_tmp.w = __shfl_xor_sync(0xffffffff, q_rot.w, 1);
        //        }
//
        //        {
        //            q_rot_tmp1.x = __shfl_sync(0xffffffff, q_rot.x, 2);
        //            q_rot_tmp1.y = __shfl_sync(0xffffffff, q_rot.y, 2);
        //            q_rot_tmp1.z = __shfl_sync(0xffffffff, q_rot.z, 2);
        //            q_rot_tmp1.w = __shfl_sync(0xffffffff, q_rot.w, 2);
        //        }
        //        if (lane %2 == 0){
        //            q_rot_tmp1.x = __shfl_xor_sync(0xffffffff, q_rot.x, 2);
        //            q_rot_tmp1.y = __shfl_xor_sync(0xffffffff, q_rot.y, 2);
        //            q_rot_tmp1.z = __shfl_xor_sync(0xffffffff, q_rot.z, 2);
        //            q_rot_tmp1.w = __shfl_xor_sync(0xffffffff, q_rot.w, 2);
        //        }
        //        q_rot.x = (lane < 1) ? q_rot_tmp.z : q_rot_tmp1.z;
        //        q_rot.y = (lane < 1) ? q_rot_tmp.w : q_rot_tmp1.w;
        //        q_rot.z = (lane > 0) ? q_rot_tmp.x : q_rot_tmp1.x;
        //        q_rot.w = (lane > 0) ? q_rot_tmp.y : q_rot_tmp1.y;
        //    }
        //    #pragma unroll
        //    for (int o = 0;o < 4;o++){
        //        int index = ((d3 << 2) + o) * 2;
        //        float inv_freq[2];
        //        inv_freq[0] = (float)((index % half_dim) * 2) / (float)(rotary_dim << 3);
        //        inv_freq[1] = (float)(((index+1) % half_dim) * 2) / (float)(rotary_dim << 3);
        //        inv_freq[0] = 1.0 / powf(10000.0, inv_freq[0]) * (float)seq_id;
        //        inv_freq[1] = 1.0 / powf(10000.0, inv_freq[1]) * (float)seq_id;
        //        q_h[o].x = (__half)((float)qrot_h[o].x * sinf(inv_freq[0]) + (float)q_h[o].x * cosf(inv_freq[0]));
        //        q_h[o].y = (__half)((float)qrot_h[o].y * sinf(inv_freq[1]) + (float)q_h[o].y * cosf(inv_freq[1]));
        //    }
        //}
        output_vec[d3] = q;
    }
    else 
        output_vec[d3] = vals_vec[d3];

#endif
}


// [B S C*H] - > C * [B A S N]
template <>
void launch_bias_add_transform_0213<float>(float* output,
                                            float* k_cache, float* v_cache,
                                           const float* vals,
                                           const float* bias,
                                           int batch_size,
                                           int seq_length,
                                           unsigned seq_offset,
                                           int all_tokens,
                                           int hidden_dim,
                                           int heads,
                                           int rotary_dim,
                                           bool rotate_half,
                                           bool rotate_every_two,
                                           hipStream_t stream,
                                           int trans_count)
{
    hidden_dim >>= 2;
    int head_ext = (hidden_dim - 1) / MAX_THREADS + 1;

    dim3 block_dim(hidden_dim / heads, (heads / head_ext));
    dim3 grid_dim(batch_size, seq_length, (trans_count * head_ext));

    bias_add_transform_0213<<<grid_dim, block_dim, 0, stream>>>(
        output, vals, bias, hidden_dim, seq_length, heads, head_ext);
}

template <>
void launch_bias_add_transform_0213<__half>(__half* output,
                                            __half* k_cache, 
                                            __half* v_cache,
                                            const __half* vals,
                                            const __half* bias,
                                            int batch_size,
                                            int seq_length,
                                            unsigned seq_offset,
                                            int all_tokens,
                                            int hidden_dim,
                                            int heads,
                                            int rotary_dim,
                                            bool rotate_half,
                                            bool rotate_every_two,
                                            hipStream_t stream,
                                            int trans_count)
{
    hidden_dim >>= 3;
    int head_ext = 1;// (hidden_dim - 1) / MAX_THREADS + 1;
    dim3 block_dim(hidden_dim / heads, (heads / head_ext));
    dim3 grid_dim(1, seq_length, (trans_count * head_ext));
    bias_add_transform_0213<<<grid_dim, block_dim, 0, stream>>>(
        output, k_cache, v_cache, vals, bias, hidden_dim, seq_length, seq_offset,
        all_tokens, heads, 
        rotary_dim >> 3,
        rotate_half,
        rotate_every_two,
        head_ext);
}



// Bias add
template <typename T>
__global__ void bias_add_transform_0213(T* output,
                                        const T* vals,
                                        const T* bias,
                                        int hidden_dim,
                                        int seq_length,
                                        int heads,
                                        int head_ext);

template <>
__global__ void bias_add_transform_0213<float>(float* output,
                                               const float* vals,
                                               const float* bias,
                                               int hidden_dim,
                                               int seq_length,
                                               int heads,
                                               int head_ext)
{
    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0_out_stride = d0_stride;
    int d1_out_stride = d2_stride;
    int d2_out_stride = d2_stride * seq_length;

    int d0 = blockIdx.x;                                                  // Batch
    int d1 = blockIdx.y;                                                  // Sequence ID (0-127)
    int cnt = blockIdx.z / head_ext;                                      // Hidden count
    int d2 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head (0-11)
    int d3 = threadIdx.x;                                                 // Values (groups of 4)

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    const float4* bias_vec = reinterpret_cast<const float4*>(bias);
    float4* output_vec = reinterpret_cast<float4*>(output);

    float4 inputs = vals_vec[d0 * d0_stride * (gridDim.z / head_ext) + cnt * d1_stride +
                             d1 * d1_stride * (gridDim.z / head_ext) + d2 * d2_stride + d3];
    float4 biases = bias_vec[cnt * d1_stride + d2 * d2_stride + d3];

    float4 outputs;
    outputs.x = inputs.x + biases.x;
    outputs.y = inputs.y + biases.y;
    outputs.z = inputs.z + biases.z;
    outputs.w = inputs.w + biases.w;

    output_vec[cnt * d0_out_stride * gridDim.x + d0 * d0_out_stride + d1 * d1_out_stride +
               d2 * d2_out_stride + d3] = outputs;
}

template <>
__global__ void bias_add_transform_0213<__half>(__half* output,
                                                const __half* vals,
                                                const __half* bias,
                                                int hidden_dim,
                                                int seq_length,
                                                int heads,
                                                int head_ext)
{
#ifdef HALF_PRECISION_AVAILABLE

    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d2_out_stride = d2_stride * seq_length;

    int d0 = blockIdx.x;                                                  // Batch
    int d1 = blockIdx.y;                                                  // Sequence ID (0-127)
    int cnt = blockIdx.z / head_ext;                                      // Hidden count
    int d2 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head (0-11)
    int d3 = threadIdx.x;                                                 // Values (groups of 4)

    float4 vals_arr;
    float4 bias_arr;
    float4 output_arr;
    __half2* vals_half = reinterpret_cast<__half2*>(&vals_arr);
    __half2* bias_half = reinterpret_cast<__half2*>(&bias_arr);
    __half2* output_half = reinterpret_cast<__half2*>(&output_arr);

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    const float4* bias_vec = reinterpret_cast<const float4*>(bias);
    float4* output_vec = reinterpret_cast<float4*>(output);

    vals_vec += (d0 * d0_stride * (gridDim.z / head_ext));
    vals_vec += (d1 * d1_stride * (gridDim.z / head_ext));
    vals_vec += (cnt * d1_stride);
    vals_vec += (d2 * d2_stride);

    bias_vec += (cnt * d1_stride);
    bias_vec += (d2 * d2_stride);

    output_vec += (cnt * d0_stride * gridDim.x);
    output_vec += (d1 * d2_stride);
    output_vec += (d0 * d0_stride);
    output_vec += (d2 * d2_out_stride);

    bias_arr = bias_vec[d3];
    vals_arr = vals_vec[d3];

    output_half[0] = vals_half[0] + bias_half[0];
    output_half[1] = vals_half[1] + bias_half[1];
    output_half[2] = vals_half[2] + bias_half[2];
    output_half[3] = vals_half[3] + bias_half[3];
    output_vec[d3] = output_arr;

#endif
}

__global__ void bias_add_transform_0213_v2(__half* output,
                                           const __half* vals,
                                           const __half* bias,
                                           int hidden_dim,
                                           int seq_length,
                                           int heads)
{
#ifdef HALF_PRECISION_AVAILABLE
    __shared__ float4 in_data[3072];

    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;
    int iteration_stride = d1_stride * blockDim.z;  // Hidden * 3 / 8
    int batch_stride = d0_stride * blockDim.z;      // Hidden * S * 3 / 8

    int d0_out_stride = d0_stride;
    int d1_out_stride = d2_stride;
    int d2_out_stride = d2_stride * seq_length;

    int d0 = blockIdx.x;    // Batch
    int d1 = blockIdx.y;    // Sequence ID (0-127)
    int cnt = threadIdx.z;  // blockIdx.z; // Hidden count
    int d2 = threadIdx.y;   // Head (0-11)
    int d3 = threadIdx.x;   // Values (groups of 4)

    float4 vals_arr[1];
    float4 bias_arr[1];
    float4 output_arr[1];
    __half2* vals_half = reinterpret_cast<__half2*>(vals_arr);
    __half2* bias_half = reinterpret_cast<__half2*>(bias_arr);
    __half2* output_half = reinterpret_cast<__half2*>(output_arr);

    const float4* vals_vec = reinterpret_cast<const float4*>(vals);
    const float4* bias_vec = reinterpret_cast<const float4*>(bias);
    float4* output_vec = reinterpret_cast<float4*>(output);

    int iter_index = cnt * d1_stride + d2 * d2_stride + d3;
    int input_offset = d0 * batch_stride + d1 * (iteration_stride << 1);
    bias_arr[0] = bias_vec[iter_index];

#pragma unroll
    for (int iter = 0; iter < 2; iter++) {
        int iter_id = iter * iteration_stride + iter_index;
        vals_arr[0] = vals_vec[input_offset + iter_id];

        output_half[0] = vals_half[0] + bias_half[0];
        output_half[1] = vals_half[1] + bias_half[1];
        output_half[2] = vals_half[2] + bias_half[2];
        output_half[3] = vals_half[3] + bias_half[3];

        in_data[iter_id] = output_arr[0];
    }
    __syncthreads();

    iteration_stride = blockDim.z * (blockDim.y >> 1);
    int matrix_stride = (d0_out_stride * gridDim.x);
    int head_count = (d2 >> 1) + cnt * (blockDim.y >> 1);

    int out_index = d0 * d0_out_stride + d1 * (d1_out_stride << 1) + d3 + (d2 % 2) * d2_stride;

#pragma unroll
    for (int iter = 0; iter < 2; iter++) {
        int iter_row = (iter * iteration_stride) + head_count;
        int iter_offset =
            (iter_row % blockDim.y) * d2_out_stride + (iter_row / blockDim.y) * matrix_stride;
        output_vec[out_index + iter_offset] =
            in_data[iter_row * d2_stride + d3 + (d2 % 2) * (d1_stride * blockDim.z)];
    }
#endif
}

// [B S C*H] - > C * [B A S N]
template <>
void launch_bias_add_transform_0213<float>(float* output,
                                           const float* vals,
                                           const float* bias,
                                           int batch_size,
                                           int seq_length,
                                           int hidden_dim,
                                           int heads,
                                           hipStream_t stream,
                                           int trans_count)
{
    hidden_dim >>= 2;
    int head_ext = (hidden_dim - 1) / MAX_THREADS + 1;

    dim3 block_dim(hidden_dim / heads, (heads / head_ext));
    dim3 grid_dim(batch_size, seq_length, (trans_count * head_ext));

    bias_add_transform_0213<float><<<grid_dim, block_dim, 0, stream>>>(
        output, vals, bias, hidden_dim, seq_length, heads, head_ext);
}

template <>
void launch_bias_add_transform_0213<__half>(__half* output,
                                            const __half* vals,
                                            const __half* bias,
                                            int batch_size,
                                            int seq_length,
                                            int hidden_dim,
                                            int heads,
                                            hipStream_t stream,
                                            int trans_count)
{
    hidden_dim >>= 3;
    if (true) { // (hidden_dim > 128 || hidden_dim < 16) {
        int head_ext = (hidden_dim - 1) / MAX_THREADS + 1;
        dim3 block_dim(hidden_dim / heads, (heads / head_ext));
        dim3 grid_dim(batch_size, seq_length, (trans_count * head_ext));
        bias_add_transform_0213<__half><<<grid_dim, block_dim, 0, stream>>>(
            output, vals, bias, hidden_dim, seq_length, heads, head_ext);
    } else {
        dim3 block_dim(hidden_dim / heads, heads, trans_count);
        dim3 grid_dim(batch_size, seq_length / 2);
        bias_add_transform_0213_v2<<<grid_dim, block_dim, 0, stream>>>(
            output, vals, bias, hidden_dim, seq_length, heads);
    }
}

template <typename T>
__global__ void transform4d_0213(T* out,
                                 const T* in,
                                 int heads,
                                 int seq_length,
                                 int hidden_dim,
                                 int head_ext);

template <>
__global__ void transform4d_0213<float>(float* out,
                                        const float* in,
                                        int heads,
                                        int seq_length,
                                        int hidden_dim,
                                        int head_ext)
{
    int d0_stride = hidden_dim * seq_length;
    int d1_stride = d0_stride / heads;
    int d2_stride = hidden_dim / heads;

    int d0_out_stride = d0_stride;
    int d1_out_stride = d2_stride;
    int d2_out_stride = hidden_dim;

    int d0 = blockIdx.x;                                        // Batch
    int d1 = blockIdx.y / ((seq_length - 1) / blockDim.y + 1);  // Head
    int d2 = (threadIdx.y + blockDim.y * blockIdx.y) % seq_length;
    int cnt = blockIdx.z;
    int d3 = threadIdx.x;  // Values (groups of 8)

    if (d2 < seq_length) {
        const float4* in_vec = reinterpret_cast<const float4*>(in);
        float4* out_vec = reinterpret_cast<float4*>(out);

        float4 vals_vec = in_vec[cnt * d0_stride * gridDim.x + d0 * d0_stride + d1 * d1_stride +
                                 d2 * d2_stride + d3];
        out_vec[d0 * d0_out_stride * gridDim.z + cnt * d2_out_stride + d1 * d1_out_stride +
                d2 * d2_out_stride * gridDim.z + d3] = vals_vec;
    }
}

template <>
__global__ void transform4d_0213<__half>(__half* out,
                                         const __half* in,
                                         int heads,
                                         int seq_length,
                                         int hidden_dim,
                                         int head_ext)
{
#if __CUDA_ARCH__ >= 700

    int d0_stride = hidden_dim * (seq_length / head_ext);
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0 = blockIdx.x;                                                  // Batch
    int d1 = threadIdx.y + (blockIdx.z % head_ext) * (heads / head_ext);  // Head
    int d2 = blockIdx.z / head_ext;                                       // Sequence
    int cnt = blockIdx.y;                                                 // Hidden count
    int d3 = threadIdx.x;                                                 // Values (groups of 8)

    const float4* in_vec = reinterpret_cast<const float4*>(in);
    float4* out_vec = reinterpret_cast<float4*>(out);

    in_vec += (cnt * d0_stride * gridDim.x);
    in_vec += (d0 * d0_stride);
    in_vec += (d2 * d2_stride);
    in_vec += (d1 * d2_stride * seq_length);

    out_vec += (cnt * d1_stride);
    out_vec += (d1 * d2_stride);
    out_vec += (d0 * d0_stride * gridDim.y);
    out_vec += (d2 * d1_stride * gridDim.y);

    out_vec[d3] = in_vec[d3];

#endif
}

__global__ void transform4d_0213_v2(__half* out,
                                    const __half* in,
                                    int heads,
                                    int seq_length,
                                    int hidden_dim)
{
#if __CUDA_ARCH__ >= 700
    __shared__ float4 in_data[3072];

    int d0_stride = hidden_dim * seq_length;
    int d1_stride = hidden_dim;
    int d2_stride = hidden_dim / heads;

    int d0 = blockIdx.x;    // Batch
    int d1 = threadIdx.y;   // Head
    int d2 = blockIdx.y;    // Sequence
    int cnt = threadIdx.z;  // Hidden count
    int d3 = threadIdx.x;   // Values (groups of 8)

    const float4* in_vec = reinterpret_cast<const float4*>(in);
    float4* out_vec = reinterpret_cast<float4*>(out);

    int input_offset = d0 * d0_stride + d2 * (d2_stride << 1) + d3 + (d1 % 2) * d2_stride;
    int head_count = (d1 >> 1) + cnt * (blockDim.y >> 1);
    int iteration_stride = blockDim.z * (blockDim.y >> 1);
    int matrix_stride = (d0_stride * gridDim.x);

#pragma unroll
    for (int iter = 0; iter < 2; iter++) {
        int iter_row = iter * iteration_stride + head_count;
        int iter_offset = (iter_row % blockDim.y) * d2_stride;

        in_data[d3 + iter_offset + (iter_row / blockDim.y + (d1 % 2) * blockDim.z) * d1_stride] =
            in_vec[input_offset + iter_offset * seq_length +
                   (iter_row / blockDim.y) * matrix_stride];
    }
    __syncthreads();

    iteration_stride = d1_stride * blockDim.z;
    int iter_index = cnt * d1_stride + d1 * d2_stride + d3;
    int output_offset = d0 * d0_stride * blockDim.z + d2 * (iteration_stride << 1);

#pragma unroll
    for (int iter = 0; iter < 2; iter++) {
        int iter_id = iter * iteration_stride + iter_index;
        out_vec[output_offset + iter_id] = in_data[iter_id];
    }
#endif
}

// 3 * [B A S N] - > [B S C*H]
template <>
void launch_transform4d_0213<float>(float* out,
                                    const float* in,
                                    int batch_size,
                                    int heads,
                                    int seq_length,
                                    int hidden_dim,
                                    hipStream_t stream,
                                    int trans_count)
{
    hidden_dim >>= 2;
    dim3 grid_dims(batch_size, heads * ((seq_length - 1) / 8 + 1), trans_count);
    dim3 block_dims(hidden_dim / heads, 8);
    transform4d_0213<float>
        <<<grid_dims, block_dims, 0, stream>>>(out, in, heads, seq_length, hidden_dim, 1);
}

template <>
void launch_transform4d_0213<__half>(__half* out,
                                     const __half* in,
                                     int batch_size,
                                     int heads,
                                     int seq_length,
                                     int hidden_dim,
                                     hipStream_t stream,
                                     int trans_count)
{
    hidden_dim >>= 3;
    int head_ext = (hidden_dim - 1) / MAX_THREADS + 1;
    dim3 grid_dims(batch_size, trans_count, (seq_length * head_ext));
    dim3 block_dims(hidden_dim / heads, (heads / head_ext));
    transform4d_0213<__half><<<grid_dims, block_dims, 0, stream>>>(
        out, in, heads, seq_length, hidden_dim, head_ext);
}
